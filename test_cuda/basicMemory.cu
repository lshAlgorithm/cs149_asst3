#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    const int N = 10;
    float *h_data, *d_data;

    // Allocate memory on the host
    h_data = new float[N];
    for (int i = 0; i < N; ++i) {
        h_data[i] = i;
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_data, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice);

    // Copy data back from device to host
    hipMemcpy(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory on the device
    hipFree(d_data);

    // Print the data on the host to verify correctness
    std::cout << "Data on the host after copying from the device:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    // Free memory on the host
    delete[] h_data;

    return 0;
}

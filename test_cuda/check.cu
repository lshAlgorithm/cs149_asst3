#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int driverVersion = 0, runtimeVersion = 0;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    std::cout << driverVersion << " " << runtimeVersion << std::endl;
    if (driverVersion / 1000 == runtimeVersion / 1000) {
        std::cout << "CUDA Driver API and Runtime API versions are compatible." << std::endl;
    } else {
        std::cout << "WARNING: CUDA Driver API and Runtime API versions are NOT COMPATIBLE." << std::endl;
    }

    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error == hipSuccess && deviceCount > 0) {
        std::cout << "CUDA is available and there are " << deviceCount << " devices." << std::endl;
        for (int i = 0; i < deviceCount; ++i) {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);
            std::cout << "Device " << i << ": " << prop.name << std::endl;
        }
    } else {
        std::cout << "No CUDA devices found or CUDA is not available." << std::endl;
    }

    return 0;
}
